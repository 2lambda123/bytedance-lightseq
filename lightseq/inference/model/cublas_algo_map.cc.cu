#pragma once

#include "cublas_algo_map.h"

namespace lightseq {
namespace cuda {}  // namespace cuda

cublasAlgoMap::cublasAlgoMap(const std::string filename)
    : _config_filename(filename) {
  loadGemmConfig();
}

cublasAlgoMap::cublasAlgoMap(const cublasAlgoMap& algo_map)
    : _config_filename(algo_map._config_filename),
      _algo_map(algo_map._algo_map),
{}

cublasAlgoMap::~cublasAlgoMap() { _algo_map.clear(); }

void cublasAlgoMap::loadGemmConfig() {
  FILE* fd;
  fd = fopen(_config_filename.c_str(), "r");
  if (fd == NULL) {
    std::cout << "[WARNING] " << _config_filename
              << " is not found; using default GEMM algo" << std::endl;
    return;
  }

  int m, n, k, algoId, customOption, tile, splitK_val, swizzle, reductionScheme,
      workspaceSize, stages, sm;
  float fp16_time, int8_time, speedup;
  char data_order[50];
  char tmp[1024];
  if (!fgets(tmp, 1024, fd)) {
    printf("[ERROR] fgets fail at %s:%d \n", __FILE__, __LINE__);
    exit(-1);
  }
  while (fscanf(fd, "%d %d %d %d %d %d %d %d %d %d %d %f %f %f %d %s\n", &m, &n,
                &k, &algoId, &tile, &splitK_val, &reductionScheme, &swizzle,
                &customOption, &stages, &workspaceSize, &fp16_time, &int8_time,
                &speedup, &sm, &data_order) != EOF) {
    std::string dataOrder(data_order);
    std::vector<int> mnk = {m, n, k};
    if (_algo_map.find(mnk) == _algo_map.end()) {
      _algo_map[mnk].algoId = algoId;
      _algo_map[mnk].customOption = customOption;
      _algo_map[mnk].tile = tile;
      _algo_map[mnk].splitK_val = splitK_val;
      _algo_map[mnk].swizzle = swizzle;
      _algo_map[mnk].reductionScheme = reductionScheme;
      _algo_map[mnk].workspaceSize = workspaceSize;
      _algo_map[mnk].stages = stages;
      _algo_map[mnk].dataOrder = dataOrder;
    }
  }
  fclose(fd);
}

bool cublasAlgoMap::isExist(const int m, const int n, const int k) {
  std::vector<int> mnk = {m, n, k};
  return _algo_map.find(mnk) != _algo_map.end();
}

cublasLtMatmulAlgo_info cublasAlgoMap::getAlgo(const int m, const int n,
                                               const int k) {
  if (m >= BORDER) m = ((m + STRIDE - 1) / STRIDE) * STRIDE;
  std::vector<int> mnk = {m, n, k};
  if (_algo_map.find(mnk) != _algo_map.end()) {
    return _algo_map[mnk];
  } else {
    cublasLtMatmulAlgo_info tmp_algo;
    tmp_algo.algoId = static_cast<int>(CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    tmp_algo.customOption = -1;
    tmp_algo.tile = -1;
    tmp_algo.splitK_val = -1;
    tmp_algo.swizzle = -1;
    tmp_algo.reductionScheme = -1;
    tmp_algo.workspaceSize = -1;
    tmp_algo.stages = -1;
    tmp_algo.dataOrder = "HIPBLASLT_ORDER_COL";
    return tmp_algo;
  }
}

}  // namespace lightseq
