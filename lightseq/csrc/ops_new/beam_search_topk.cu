#include "hip/hip_runtime.h"
#include "beam_search_topk.h"

namespace lightseq {

template <typename T>
std::tuple<Variable*, Variable*, Variable*> BeamSearchTopkOp<T>::operator()(
    Variable* logits, Variable* logit_bias, Variable* seq_probs,
    Variable* seq_score, Variable* alive_seq) {
  Variable* can_idx = new Variable(
      "can_idx", _max_batch_size * _beam_size * _trg_vocab_size * sizeof(int));

  Variable* can_score =
      new Variable("can_score", _max_batch_size * _beam_size * _trg_vocab_size *
                                    sizeof(float));

  Variable* num_beam_can = new Variable(
      "num_beam_can", (_max_batch_size * _beam_size + 1) * sizeof(int));

  this->set_parents({logits, logit_bias, seq_probs, seq_score, alive_seq});
  this->set_children({can_idx, can_score, num_beam_can});
  return std::make_tuple(can_idx, can_score, num_beam_can);
}

template <typename T>
void BeamSearchTopkOp<T>::forward() {
  hipStream_t stream = _context_ptr->get_stream();
  T* logits_ptr = (T*)parent(0)->value();
  T* logits_bias_ptr = (T*)parent(1)->value();
  float* seq_probs_ptr = (float*)parent(2)->value();
  float* seq_score_ptr = (float*)parent(3)->value();
  int* alive_seq_ptr = (int*)parent(4)->value();

  int* can_idx_ptr = (int*)child(0)->value();
  float* can_score_ptr = (float*)child(1)->value();
  int* num_beam_can_ptr = (int*)child(2)->value();

  /*
    step 1. logits bias and softmax,
      select rough topk candidate for every batch item,
      record the candidate's beam_id, vocab_id and probability
  */

  hipMemsetAsync(num_beam_can_ptr, 0, sizeof(int), stream);

  select_beam_rough_topk_launcher(
      logits_ptr, logits_bias_ptr, seq_probs_ptr, seq_score_ptr, alive_seq_ptr,
      can_idx_ptr, can_score_ptr, num_beam_can_ptr, _trg_vocab_size, _max_step,
      _length_norm, _cur_step, _step_token_num, _max_thread_per_block, stream,
      _beam_size, _diverse_lambda, _end_id);

  thrust::exclusive_scan(thrust::cuda::par.on(stream), num_beam_can_ptr + 1,
                         num_beam_can_ptr + 1 + _step_token_num,
                         num_beam_can_ptr + 1);

  /* ---step 2. sort the candidate with their probability--- */
  CHECK_GPU_ERROR(hipMemcpyAsync(&_host_can_num_batch, num_beam_can_ptr,
                                  sizeof(int), hipMemcpyDeviceToHost, stream));
  CHECK_GPU_ERROR(hipStreamSynchronize(stream));

  if (_diverse_lambda != 0) {
    if (_host_can_num_batch < _cub_sort_buffer_bytes / 160) {
      CHECK_GPU_ERROR(hipcub::DeviceRadixSort::SortPairsDescending(
          (void*)logits_ptr, _cub_sort_buffer_bytes, can_score_ptr,
          can_score_ptr, can_idx_ptr, can_idx_ptr, _host_can_num_batch, 0,
          sizeof(float) * 8, stream));
    } else {
      thrust::sort_by_key(thrust::cuda::par.on(stream), can_score_ptr,
                          can_score_ptr + _host_can_num_batch, can_idx_ptr,
                          thrust::greater<float>());
    }
    ker_diverse_beam_search_launcher(can_score_ptr, can_idx_ptr,
                                     num_beam_can_ptr, _step_token_num,
                                     _max_thread_per_block, stream, _beam_size,
                                     _diverse_lambda, _trg_vocab_size);
  }

  thrust::sort_by_key(thrust::cuda::par.on(stream), can_score_ptr,
                      can_score_ptr + _host_can_num_batch, can_idx_ptr,
                      thrust::greater<float>());
}

template class BeamSearchTopkOp<float>;
template class BeamSearchTopkOp<__half>;

}  // namespace lightseq
